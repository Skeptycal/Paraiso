#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <thrust/host_vector.h>
#include <unistd.h>

#include "Hydro.hpp"

using namespace std;

const int antiAlias = 1;
int W,H;

typedef double Real;

void dump (string fn, Hydro &sim) {
  ofstream ofs (fn.c_str()); 
  thrust::host_vector<Real> dens, vx, vy, p;

  dens = sim.static_7_density;
  vx = sim.static_8_velocity0;
  vy = sim.static_9_velocity1;
  p = sim.static_10_pressure;

  for (int iy = antiAlias/2; iy < H; iy+=antiAlias) {
    for (int ix = antiAlias/2; ix < W; ix+=antiAlias) {
      double x = sim.static_3_dR0 * (ix+0.5);
      double y = sim.static_4_dR1 * (iy+0.5);
      int i = sim.memorySize0() * (iy+sim.lowerMargin1()) + ix + sim.lowerMargin0();
      ofs << x << " " << y << " "
          << dens[i] << " "
          << vx[i] << " "
          << vy[i] << " "
          << p[i] << endl;
    }
    ofs << endl;
  }
}

struct Field {
  virtual void at(const double t, const double x, const double y,
		  double &dens, double &vx, double &vy, double &p) = 0;
};


struct EntropyWave : public Field {
  virtual void at(const double t, const double x, const double y,
	  double &dens, double &vx, double &vy, double &p) {
    vx = 1;
    vy = 0;
    dens = 2 + sin(6.2832*(x - vx * t));
    p = 1;
  }
};

struct SoundWave : public Field {
  virtual void at(const double t, const double x, const double y,
	  double &dens, double &vx, double &vy, double &p) {
    const Real kGamma = 5.0 / 3.0;
    const Real soundSpeed = 1.0;
    const Real amplitude = 1e-3;

    const Real dens0 = kGamma;
    const Real p0 = 1;
    
    vx = amplitude * sin(6.2832*(x - soundSpeed * t));
    vy = 0;
    dens = dens0 + dens0/soundSpeed * vx;
    p = p0 + kGamma * p0 / soundSpeed * vx;
  }
};




void override (double t, Field &solution, Hydro &sim) {
  thrust::host_vector<Real> dens, vx, vy, p;

  const int iR = 10;
  
  dens = sim.static_7_density;
  vx   = sim.static_8_velocity0;
  vy   = sim.static_9_velocity1;
  p    = sim.static_10_pressure;

  for (int iy = 0; iy < H; ++iy) {
    for (int ix = 0; ix < W; ++ix) {
      double x = sim.static_3_dR0 * (ix+0.5);
      double y = sim.static_4_dR1 * (iy+0.5);
      int i = sim.memorySize0() * (iy+sim.lowerMargin1()) + ix + sim.lowerMargin0();
      
      if (t < 0.1 || ix < iR || iy < iR || ix >= W-iR || iy >= H-iR) {
	Real dens0, vx0, vy0, p0;
	solution.at(t,x,y,  dens0, vx0, vy0, p0);
	
	dens[i] = dens0;
	vx[i] =  vx0;
	vy[i] =  vy0;
	p[i]  =  p0;
      }
    }
  }

  sim.static_7_density   = dens ; 
  sim.static_8_velocity0 = vx   ;
  sim.static_9_velocity1 = vy   ;
  sim.static_10_pressure = p    ;

}



int main () {
  hipSetDevice(2);
  Hydro sim;
  W = sim.size0();
  H = sim.size1();

  sim.static_1_time = 0;
  sim.static_2_cfl = 0.5;
  sim.static_5_extent0 = 1.0;
  sim.static_6_extent1 = 1.0;
  sim.static_3_dR0 = sim.static_5_extent0 / W;
  sim.static_4_dR1 = sim.static_6_extent1 / H;
  sim.init();
  char buf[256];
  sprintf(buf, "mkdir -p output-g%d", antiAlias);
  system(buf);
  int ctr = 0;

  SoundWave f;
  
  while (ctr <= 10) {
    double t = sim.static_1_time;
    cerr << sim.static_1_time << endl;
    if (!isfinite(t)) return -1;
    override(t, f, sim);
    sim.proceed();
    if (t > 0.1 * ctr) {
      sprintf(buf, "output-g%d/snapshot%04d.txt", antiAlias, ctr);
      dump(buf, sim);
      ++ctr;
    }
  }
}
